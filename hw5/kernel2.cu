#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__device__ int diverge_gpu(float c_re, float c_im, int max) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < max; ++i) {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(int *c, float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
	/*
	int pix_per_thread = resX * resY / (gridDim.x * blockDim.x);
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = pix_per_thread * tId;

	int i;
	for (i = offset; i < offset + pix_per_thread; i++){
		int x = i % resX;
		int y = i / resX;
		float cr = lowerX + x * stepX;
		float ci = lowerY + y * stepY;
		c[y * resX + x] = diverge_gpu(cr, ci, maxIterations);
	}

	if (gridDim.x * blockDim.x * pix_per_thread < resX * resY && tId < (resX * resY) - (blockDim.x * gridDim.x)){
		int i = blockDim.x * gridDim.x * pix_per_thread + tId;
		int x = i % resX;
		int y = i / resX;
		float cr = lowerX + x * stepX;
		float ci = lowerY + y * stepY;
		c[y * resY + x] = diverge_gpu(cr, ci, maxIterations);
	}
	*/
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	int thisX = index % resX;
	int thisY = index / resX;

	//int *partc = (int *) ((char *) c + thisX * pitch);

	float x = lowerX + thisX * stepX;
	float y = lowerY + thisY * stepY;
	c[index] = diverge_gpu(x, y, maxIterations);

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

	int size = resX * resY;

	int *h_c;
	int *d_c;
	size_t pitch;
	//h_c = (int *)malloc(size * sizeof(int));
	hipHostAlloc(&h_c, sizeof(int)*size, hipHostMallocMapped);
	hipHostGetDevicePointer(&d_c, h_c, 0);
	//cudaMalloc(&d_c, size * sizeof(int));
	hipMallocPitch((void **)&d_c, &pitch, sizeof(int)*resX, resY);
	//cudaMemcpy2D(d_c, pitch, h_c, sizeof(int)*resX, sizeof(int)*resX, resY, cudaMemcpyHostToDevice);

	int block_size = 1024;
	int grid_size = 1;

	//dim3 dimBlock(block_size, block_size);
	//dim3 dimGrid(grid_size, grid_size);

	mandelKernel<<<size/block_size, block_size>>>(d_c, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations);

	hipDeviceSynchronize();

	//cudaMemcpy2D(h_c, sizeof(int)*resX, d_c, pitch, sizeof(int)*resX, resY, cudaMemcpyDeviceToHost);
	hipMemcpy(h_c, d_c, size*sizeof(int), hipMemcpyDeviceToHost);

	memcpy(img, h_c, size * sizeof(int));

	hipFree(d_c);
	//free(h_c);
	hipHostFree(h_c);
}